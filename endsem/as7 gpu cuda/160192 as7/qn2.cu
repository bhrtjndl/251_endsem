
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define NUM 10000000

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))


__device__ void function(int *a,int j)
{
    //double square = a ->num1 * a->num1 +  a->num2 * a->num2  + 2 * a->num1 * a->num2;
    //a->result = log(square)/sin(square);
    *(a) = (*a)^(*(a + j/2));
    
    return;
}
__global__ void calculate(int *mem, int num,int j)
{
      int i = blockDim.x * blockIdx.x + threadIdx.x;
      if((i + j/2) >= num)
           return;
      if(i%j == 0){
      	int *a = (int *)(mem + i);
      	function(a,j);
      }
      else
      	return;
}

int main(int argc, char **argv)
{
    struct timeval start, end, t_start, t_end;
    int i;
    int *ptr;
    int *gpu_mem;   
    unsigned long num = NUM;   /*Default value of num from MACRO*/
    int blocks;
    int seed;

    if(argc == 3){
         num = atoi(argv[1]);   /*Update after checking*/
         if(num <= 0)
               num = NUM;
         seed = atoi(argv[2]);
    }
    else{
    	printf("not correct input");
    	return -1;
    }

    /* Allocate host (CPU) memory and initialize*/

    ptr = (int *)malloc(num * sizeof(int));
    
    srand(seed);
    for(i=0; i<num; ++i){
       ptr[i] = random();
    }
    
    
    gettimeofday(&t_start, NULL);
    
    /* Allocate GPU memory and copy from CPU --> GPU*/

    hipMalloc(&gpu_mem, num * sizeof(int));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(gpu_mem, ptr, num * sizeof(int) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");
    
    gettimeofday(&start, NULL);
    
    blocks = num /1024;
    
    if(num % 1024)
           ++blocks;
	int j;
	for(j=2 ; j<2*num ; j*=2){
    	calculate<<<blocks, 1024>>>(gpu_mem, num,j);
    	CUDA_ERROR_EXIT("kernel invocation");
    }
    gettimeofday(&end, NULL);
    
    /* Copy back result*/

    hipMemcpy(ptr, gpu_mem, num * sizeof(int) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);
    
    printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(gpu_mem);
   
    /*Print the last element for sanity check*/ 
    //pa = (struct num_array *) (sptr + (num -1)*3*sizeof(double));
    printf("result = %d\n",*(ptr));

    
    free(ptr);
}
